#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include "Header.h"

hipError_t refreshPointsMemoryWithCuda(Point *points, unsigned int n, double time);

__global__ void refreshPointsKernel(Point *points, int n, double time)
{
    int i = blockIdx.x * NUM_THREADS_IN_BLOCK + threadIdx.x;
	if (i < n)
	{
		points[i].x = points[i].x + time * points[i].vx;
		points[i].y = points[i].y + time * points[i].vy;
	}
}

int cudaRefreshPoints(Point* points, int n, double time)
{
    // Add vectors in parallel.
    hipError_t cudaStatus = refreshPointsMemoryWithCuda(points, n, time);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "refreshPointsMemoryWithCuda failed!");
        return 1;
    }

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t refreshPointsMemoryWithCuda(Point *points, unsigned int n, double time)
{
    Point *dev_points = 0;
    hipError_t cudaStatus;

	int numBlocks;
	if (n % NUM_THREADS_IN_BLOCK == 0)
		numBlocks = n / NUM_THREADS_IN_BLOCK;
	else
		numBlocks = (n / NUM_THREADS_IN_BLOCK) + 1;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return cudaStatus;
    }
	
    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_points, n * sizeof(Point));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
		freeResources(1, dev_points);
		return cudaStatus;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_points, points, n * sizeof(Point), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
		freeResources(1, dev_points);
		return cudaStatus;
    }
	
    // Launch a kernel on the GPU with one thread for each element.
    refreshPointsKernel <<<numBlocks, NUM_THREADS_IN_BLOCK>>>(dev_points, n, time);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		freeResources(1, dev_points);
		return cudaStatus;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		freeResources(1, dev_points);
		return cudaStatus;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(points, dev_points, n * sizeof(Point), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
		freeResources(1, dev_points);
		return cudaStatus;
    }

	freeResources(1, dev_points);
    return cudaStatus;
}