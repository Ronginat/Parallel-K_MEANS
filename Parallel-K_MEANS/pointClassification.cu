#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include "Header.h"

hipError_t classifiedPointsMemoryWithCuda(Cluster* clusters, Point *points, unsigned int n, unsigned int k, bool* isPointChangedCluster);

__device__ double Distance(double x1, double y1, double x2, double y2)
{
	return sqrt(pow(x1 - x2, 2) + pow(y1 - y2, 2));
}

__global__ void classifiedPointsKernel(Cluster* clusters, Point *points, bool *flags, unsigned int k, unsigned int n)
{
	int idx = blockIdx.x * NUM_THREADS_IN_BLOCK + threadIdx.x;
	if (idx < n)
	{
		int minIndex = -1;
		double minDistance = DBL_MAX;
		for (int i = 0; i < k; i++)
		{
			double distanceTmp = Distance(points[idx].x, points[idx].y, clusters[i].centerX, clusters[i].centerY);
			if (distanceTmp < minDistance)
			{
				minDistance = distanceTmp;
				minIndex = i;
			}
		}
		if (points[idx].clusterID != minIndex)
			flags[idx] = true;

		points[idx].clusterID = minIndex;
	}
}


int cudaClassifiedPoints(Cluster* clusters, Point* points, int n, int k, bool* isPointChangedCluster)
{
	*isPointChangedCluster = false;
	hipError_t cudaStatus = classifiedPointsMemoryWithCuda(clusters, points, n, k, isPointChangedCluster);

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "groupPointsMemoryWithCuda failed!");
		return 1;
	}

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t classifiedPointsMemoryWithCuda(Cluster* clusters, Point *points, unsigned int n, unsigned int k, bool* isPointChangedCluster)
{
	Cluster *dev_clusters;
	Point *dev_points;
	hipError_t cudaStatus;
	bool* dev_flags;
	bool* flags = (bool*)malloc(n * sizeof(bool));
	int numBlocks;

	initFlagsArrOmp(&flags, n);
	/*for (int i = 0; i < n; i++)
	flags[i] = false;*/

	//dim3 dimGrid(numBlocks, 4, 4);
	if (n % NUM_THREADS_IN_BLOCK == 0)
		numBlocks = n / NUM_THREADS_IN_BLOCK;
	else
		numBlocks = (n / NUM_THREADS_IN_BLOCK) + 1;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return cudaStatus;
	}
	
	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_clusters, k * sizeof(Cluster));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc clusters failed!");
		freeResources(1, dev_clusters);
		return cudaStatus;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_points, n * sizeof(Point));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc points failed!");
		freeResources(2, dev_points, dev_clusters);
		return cudaStatus;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_flags, n * sizeof(bool));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc flags failed!");
		freeResources(3, dev_points, dev_clusters, dev_flags);
		return cudaStatus;
	}
	
	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_points, points, n * sizeof(Point), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy points failed!");
		freeResources(3, dev_points, dev_clusters, dev_flags);
		return cudaStatus;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_clusters, clusters, k * sizeof(Cluster), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		freeResources(3, dev_points, dev_clusters, dev_flags);
		return cudaStatus;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_flags, flags, n * sizeof(bool), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy flags failed!");
		freeResources(3, dev_points, dev_clusters, dev_flags);
		return cudaStatus;
	}
	// Launch a kernel on the GPU with one thread for each point.
	classifiedPointsKernel << <numBlocks, NUM_THREADS_IN_BLOCK >> >(dev_clusters, dev_points, dev_flags, k, n);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "classifiedPointsKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		freeResources(3, dev_points, dev_clusters, dev_flags);
		return cudaStatus;
	}
	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching classifiedPointsKernel!\n", cudaStatus);
		freeResources(3, dev_points, dev_clusters, dev_flags);
		return cudaStatus;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(points, dev_points, n * sizeof(Point), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy points failed!");
		freeResources(3, dev_points, dev_clusters, dev_flags);
		return cudaStatus;
	}

	cudaStatus = hipMemcpy(flags, dev_flags, n * sizeof(bool), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy flags failed!");
		freeResources(3, dev_points, dev_clusters, dev_flags);
		return cudaStatus;
	}

	*isPointChangedCluster = mergeFlagsArrOmp(flags, n);

	free(flags);
	freeResources(3, dev_points, dev_clusters, dev_flags);

	return cudaStatus;
}

int freeResources(int size, ...)
{
	hipError_t cudaStatus;
	va_list list;
	va_start(list, size);
	for (int i = 0; i < size; i++)
	{
		cudaStatus = hipFree(va_arg(list, void*));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipFree failed!");
			return 1;
		}
	}
	va_end(list);
	return 0;
}